#include "hip/hip_runtime.h"
#include "dark_cuda.h"

#include <stdio.h>
#include <time.h>
#include <assert.h>

#include "network.h"
#include "image.h"
#include "data.h"
#include "utils.h"
#include "parser.h"

#include "crop_layer.h"
#include "connected_layer.h"
#include "rnn_layer.h"
#include "gru_layer.h"
#include "crnn_layer.h"
#include "detection_layer.h"
#include "region_layer.h"
#include "convolutional_layer.h"
#include "activation_layer.h"
#include "maxpool_layer.h"
#include "reorg_layer.h"
#include "avgpool_layer.h"
#include "normalization_layer.h"
#include "batchnorm_layer.h"
#include "cost_layer.h"
#include "local_layer.h"
#include "softmax_layer.h"
#include "dropout_layer.h"
#include "route_layer.h"
#include "shortcut_layer.h"
#include "blas.h"

//#ifdef OPENCV
//#include <opencv2/highgui/highgui_c.h>
//#endif

#include "http_stream.h"

float* get_network_output_gpu_layer(network net, int i);
float* get_network_delta_gpu_layer(network net, int i);
float* get_network_output_gpu(network net);


void forward_network_gpu(network net, network_state state)
{

    //printf("\n");
    state.workspace = net.workspace;
    int i;
    for (i = 0; i < net.n; ++i) {
        state.index = i;
        layer l = net.layers[i];
        if (l.delta_gpu && state.train) {
            fill_ongpu(l.outputs * l.batch, 0, l.delta_gpu, 1);
        }

        l.forward_gpu(l, state);

        if (net.wait_stream)
            hipStreamSynchronize(get_cuda_stream());
        state.input = l.output_gpu;
    }

}

void backward_network_gpu(network net, network_state state)
{
    state.workspace = net.workspace;
    int i;
    float* original_input = state.input;
    float* original_delta = state.delta;
    for (i = net.n - 1; i >= 0; --i) {
        state.index = i;
        layer l = net.layers[i];
        if (l.stopbackward == 1) break;
        if (l.stopbackward > get_current_iteration(net)) break;
        if (i == 0) {
            state.input = original_input;
            state.delta = original_delta;
        }
        else {
            layer prev = net.layers[i - 1];
            state.input = prev.output_gpu;
            state.delta = prev.delta_gpu;
            if (net.optimized_memory && !prev.keep_delta_gpu) {
                state.delta = net.state_delta_gpu;
            }
        }
        if (l.onlyforward) continue;

        l.backward_gpu(l, state);

        if (i != 0) {
            layer prev = net.layers[i - 1];
            if (net.optimized_memory && state.delta && !prev.keep_delta_gpu) {
                if (prev.delta_gpu != state.delta) simple_copy_ongpu(prev.outputs * prev.batch, state.delta, prev.delta_gpu);
                fill_ongpu(prev.outputs * prev.batch, 0, net.state_delta_gpu, 1);
            }
        }

        /*
        if(i != 0)
        {
            layer l = net.layers[i - 1];
            int state_delta_nan_inf = is_nan_or_inf(state.delta, l.outputs * l.batch);
            int state_input_nan_inf = is_nan_or_inf(state.input, l.outputs * l.batch);
            printf("\n i - %d  is_nan_or_inf(s.delta) = %d \n", i, state_delta_nan_inf);
            printf(" i - %d  is_nan_or_inf(s.input) = %d \n", i, state_input_nan_inf);
            if (state_delta_nan_inf || state_input_nan_inf) { printf(" found "); getchar(); }
        }
        */
    }

    if (net.adversarial && net.attention)
    {
        int img_size = net.w * net.h * net.c;
        float* original_input_cpu = (float*)xcalloc(img_size, sizeof(float));
        float* original_delta_cpu = (float*)xcalloc(img_size, sizeof(float));
        cuda_pull_array(original_input, original_input_cpu, img_size);
        cuda_pull_array(original_delta, original_delta_cpu, img_size);

        image attention_img = make_attention_image(img_size, original_delta_cpu, original_input_cpu, net.w, net.h, net.c);
        show_image(attention_img, "attention_img");
        resize_window_cv("attention_img", 500, 500);

        free_image(attention_img);

        free(original_input_cpu);
        free(original_delta_cpu);
    }
    if (net.adversarial) {
        int x_size = get_network_input_size(net) * net.batch;
        printf(" x_size = %d, original_delta = %p, original_input = %p, net.learning_rate = %f \n",
            x_size, original_delta, original_input, net.learning_rate);
        axpy_ongpu(x_size, net.learning_rate, original_delta, 1, original_input, 1);
        constrain_min_max_ongpu(x_size, 0, 1, original_input, 1);
    }
}

void update_network_gpu(network net)
{
    cuda_set_device(net.gpu_index);
    const int iteration_num = (*net.seen) / (net.batch * net.subdivisions);
    int i;
    int update_batch = net.batch * net.subdivisions * get_sequence_value(net);
    float rate = get_current_rate(net);
    for (i = 0; i < net.n; ++i) {
        layer l = net.layers[i];
        l.t = get_current_batch(net);
        if (iteration_num > (net.max_batches * 1 / 2)) l.deform = 0;
        if (l.burnin_update && (l.burnin_update * net.burn_in > iteration_num)) continue;
        if (l.train_only_bn) continue;

        if (l.update_gpu && l.dont_update < iteration_num) {
            l.update_gpu(l, update_batch, rate, net.momentum, net.decay, net.loss_scale);
        }
    }
}

void forward_backward_network_gpu(network net, float* x, float* y)
{
    network_state state;
    state.index = 0;
    state.net = net;
    int x_size = get_network_input_size(net) * net.batch;
    int y_size = get_network_output_size(net) * net.batch;
    if (net.layers[net.n - 1].truths) y_size = net.layers[net.n - 1].truths * net.batch;
    if (!*net.input_gpu) {
        *net.input_gpu = cuda_make_array(x, x_size);
        *net.truth_gpu = cuda_make_array(y, y_size);
    }
    else {
        cuda_push_array(*net.input_gpu, x, x_size);
        cuda_push_array(*net.truth_gpu, y, y_size);
    }
    state.input = *net.input_gpu;
    state.delta = 0;
    if (net.adversarial) {
        state.delta = cuda_make_array(NULL, x_size);
    }
    state.truth = *net.truth_gpu;
    state.train = 1;
#if defined(CUDNN_HALF) && defined(CUDNN)
    int i;
    for (i = 0; i < net.n; ++i) {
        layer l = net.layers[i];
        if (net.cudnn_half) {
            if (l.type == CONVOLUTIONAL && l.weights_gpu && l.weights_gpu16) {
                assert((l.nweights) > 0);
                cuda_convert_f32_to_f16(l.weights_gpu, l.nweights, l.weights_gpu16);
            }
            else if (l.type == CRNN && l.input_layer->weights_gpu && l.input_layer->weights_gpu16) {
                assert((l.input_layer->c * l.input_layer->n * l.input_layer->size * l.input_layer->size) > 0);
                cuda_convert_f32_to_f16(l.input_layer->weights_gpu, l.input_layer->nweights, l.input_layer->weights_gpu16);
                cuda_convert_f32_to_f16(l.self_layer->weights_gpu, l.self_layer->nweights, l.self_layer->weights_gpu16);
                cuda_convert_f32_to_f16(l.output_layer->weights_gpu, l.output_layer->nweights, l.output_layer->weights_gpu16);
            }
            else if (l.type == CONV_LSTM && l.wf->weights_gpu && l.wf->weights_gpu16) {
                assert((l.wf->c * l.wf->n * l.wf->size * l.wf->size) > 0);
                if (l.peephole) {
                    cuda_convert_f32_to_f16(l.vf->weights_gpu, l.vf->nweights, l.vf->weights_gpu16);
                    cuda_convert_f32_to_f16(l.vi->weights_gpu, l.vi->nweights, l.vi->weights_gpu16);
                    cuda_convert_f32_to_f16(l.vo->weights_gpu, l.vo->nweights, l.vo->weights_gpu16);
                }
                cuda_convert_f32_to_f16(l.wf->weights_gpu, l.wf->nweights, l.wf->weights_gpu16);
                if (!l.bottleneck) {
                    cuda_convert_f32_to_f16(l.wi->weights_gpu, l.wi->nweights, l.wi->weights_gpu16);
                    cuda_convert_f32_to_f16(l.wg->weights_gpu, l.wg->nweights, l.wg->weights_gpu16);
                    cuda_convert_f32_to_f16(l.wo->weights_gpu, l.wo->nweights, l.wo->weights_gpu16);
                }
                cuda_convert_f32_to_f16(l.uf->weights_gpu, l.uf->nweights, l.uf->weights_gpu16);
                cuda_convert_f32_to_f16(l.ui->weights_gpu, l.ui->nweights, l.ui->weights_gpu16);
                cuda_convert_f32_to_f16(l.ug->weights_gpu, l.ug->nweights, l.ug->weights_gpu16);
                cuda_convert_f32_to_f16(l.uo->weights_gpu, l.uo->nweights, l.uo->weights_gpu16);
            }
        }
    }
#endif
    forward_network_gpu(net, state);
    //hipStreamSynchronize(get_cuda_stream());
    backward_network_gpu(net, state);

    if (net.adversarial) {
        cuda_free(state.delta);
        cuda_pull_array(*net.input_gpu, x, x_size);
    }
    if (*(state.net.total_bbox) > 0)
        fprintf(stderr, " total_bbox = %d, rewritten_bbox = %f %% \n", *(state.net.total_bbox), 100 * (float)*(state.net.rewritten_bbox) / *(state.net.total_bbox));
}

float train_network_datum_gpu(network net, float* x, float* y)
{
    *net.seen += net.batch;
    if (net.adversarial_lr && rand_int(0, 1) == 1 && get_current_iteration(net) > net.burn_in) {
        net.adversarial = 1;
        float lr_old = net.learning_rate;
        float scale = (get_current_iteration(net) / ((float)net.max_batches));
        //scale = sin(scale * M_PI);
        net.learning_rate = net.adversarial_lr * scale;
        layer l = net.layers[net.n - 1];
        int y_size = get_network_output_size(net) * net.batch;
        if (net.layers[net.n - 1].truths) y_size = net.layers[net.n - 1].truths * net.batch;
        float* truth_cpu = (float*)xcalloc(y_size, sizeof(float));

        const int img_size = net.w * net.h * net.c;
        float* old_input = (float*)xcalloc(img_size * net.batch, sizeof(float));
        memcpy(old_input, x, img_size * net.batch * sizeof(float));

        printf("\n adversarial training, adversarial_lr = %f \n", net.adversarial_lr * scale);

        forward_backward_network_gpu(net, x, truth_cpu);

        int b;
        for (b = 0; b < net.batch; ++b) {
            if (b % 2 == 1 && net.contrastive) {
                //printf(" b = %d old img, ", b);
                memcpy(x + img_size * b, old_input + img_size * b, img_size * sizeof(float));
            }
        }

        image im;
        im.w = net.w;
        im.h = net.h;
        im.c = net.c;
        im.data = x;
        show_image(im, "adversarial data augmentation");
        resize_window_cv("adversarial data augmentation", 500, 500);
        wait_key_cv(1);

        free(old_input);
        free(truth_cpu);
        net.learning_rate = lr_old;
        net.adversarial = 0;
    }
    forward_backward_network_gpu(net, x, y);
    float error = get_network_cost(net);
    //if (((*net.seen) / net.batch) % net.subdivisions == 0) update_network_gpu(net);
    const int sequence = get_sequence_value(net);
    //if (((*net.seen) / net.batch) % (net.subdivisions*sequence) == 0) update_network_gpu(net);

    return error;
}

typedef struct {
    network net;
    data d;
    float* err;
} train_args;

void* train_thread(void* ptr)
{
    train_args args = *(train_args*)ptr;
    free(ptr);
    cuda_set_device(args.net.gpu_index);
    *args.err = train_network(args.net, args.d);
    return 0;
}


void pull_updates(layer l)
{
    if (l.type == CONVOLUTIONAL) {
        cuda_pull_array(l.bias_updates_gpu, l.bias_updates, l.n);
        cuda_pull_array(l.weight_updates_gpu, l.weight_updates, l.nweights);
        if (l.scale_updates) cuda_pull_array(l.scale_updates_gpu, l.scale_updates, l.n);
    }
    else if (l.type == CONNECTED) {
        cuda_pull_array(l.bias_updates_gpu, l.bias_updates, l.outputs);
        cuda_pull_array(l.weight_updates_gpu, l.weight_updates, l.outputs * l.inputs);
    }
}

void push_updates(layer l)
{
    if (l.type == CONVOLUTIONAL) {
        cuda_push_array(l.bias_updates_gpu, l.bias_updates, l.n);
        cuda_push_array(l.weight_updates_gpu, l.weight_updates, l.nweights);
        if (l.scale_updates) cuda_push_array(l.scale_updates_gpu, l.scale_updates, l.n);
    }
    else if (l.type == CONNECTED) {
        cuda_push_array(l.bias_updates_gpu, l.bias_updates, l.outputs);
        cuda_push_array(l.weight_updates_gpu, l.weight_updates, l.outputs * l.inputs);
    }
}

void update_layer(layer l, network net)
{
    int update_batch = net.batch * net.subdivisions;
    float rate = get_current_rate(net);
    l.t = get_current_batch(net);
    if (l.update_gpu) {
        l.update_gpu(l, update_batch, rate, net.momentum, net.decay, net.loss_scale);
    }
}

void merge_weights(layer l, layer base)
{
    if (l.type == CONVOLUTIONAL) {
        axpy_cpu(l.n, 1, l.biases, 1, base.biases, 1);
        axpy_cpu(l.nweights, 1, l.weights, 1, base.weights, 1);
        if (l.scales) {
            axpy_cpu(l.n, 1, l.scales, 1, base.scales, 1);
        }
    }
    else if (l.type == CONNECTED) {
        axpy_cpu(l.outputs, 1, l.biases, 1, base.biases, 1);
        axpy_cpu(l.outputs * l.inputs, 1, l.weights, 1, base.weights, 1);
    }
}

void scale_weights(layer l, float s)
{
    if (l.type == CONVOLUTIONAL) {
        scal_cpu(l.n, s, l.biases, 1);
        scal_cpu(l.nweights, s, l.weights, 1);
        if (l.scales) {
            scal_cpu(l.n, s, l.scales, 1);
        }
    }
    else if (l.type == CONNECTED) {
        scal_cpu(l.outputs, s, l.biases, 1);
        scal_cpu(l.outputs * l.inputs, s, l.weights, 1);
    }
}


void pull_weights(layer l)
{
    if (l.type == CONVOLUTIONAL) {
        cuda_pull_array(l.biases_gpu, l.biases, l.n);
        cuda_pull_array(l.weights_gpu, l.weights, l.nweights);
        if (l.scales) cuda_pull_array(l.scales_gpu, l.scales, l.n);
    }
    else if (l.type == CONNECTED) {
        cuda_pull_array(l.biases_gpu, l.biases, l.outputs);
        cuda_pull_array(l.weights_gpu, l.weights, l.outputs * l.inputs);
    }
}

void push_weights(layer l)
{
    if (l.type == CONVOLUTIONAL) {
        cuda_push_array(l.biases_gpu, l.biases, l.n);
        cuda_push_array(l.weights_gpu, l.weights, l.nweights);
        if (l.scales) cuda_push_array(l.scales_gpu, l.scales, l.n);
    }
    else if (l.type == CONNECTED) {
        cuda_push_array(l.biases_gpu, l.biases, l.outputs);
        cuda_push_array(l.weights_gpu, l.weights, l.outputs * l.inputs);
    }
}

void distribute_weights(layer l, layer base)
{
    if (l.type == CONVOLUTIONAL) {
        cuda_push_array(l.biases_gpu, base.biases, l.n);
        cuda_push_array(l.weights_gpu, base.weights, l.nweights);
        if (base.scales) cuda_push_array(l.scales_gpu, base.scales, l.n);
    }
    else if (l.type == CONNECTED) {
        cuda_push_array(l.biases_gpu, base.biases, l.outputs);
        cuda_push_array(l.weights_gpu, base.weights, l.outputs * l.inputs);
    }
}


void merge_updates(layer l, layer base)
{
    if (l.type == CONVOLUTIONAL) {
        axpy_cpu(l.n, 1, l.bias_updates, 1, base.bias_updates, 1);
        axpy_cpu(l.nweights, 1, l.weight_updates, 1, base.weight_updates, 1);
        if (l.scale_updates) {
            axpy_cpu(l.n, 1, l.scale_updates, 1, base.scale_updates, 1);
        }
    }
    else if (l.type == CONNECTED) {
        axpy_cpu(l.outputs, 1, l.bias_updates, 1, base.bias_updates, 1);
        axpy_cpu(l.outputs * l.inputs, 1, l.weight_updates, 1, base.weight_updates, 1);
    }
}

void distribute_updates(layer l, layer base)
{
    if (l.type == CONVOLUTIONAL) {
        cuda_push_array(l.bias_updates_gpu, base.bias_updates, l.n);
        cuda_push_array(l.weight_updates_gpu, base.weight_updates, l.nweights);
        if (base.scale_updates) cuda_push_array(l.scale_updates_gpu, base.scale_updates, l.n);
    }
    else if (l.type == CONNECTED) {
        cuda_push_array(l.bias_updates_gpu, base.bias_updates, l.outputs);
        cuda_push_array(l.weight_updates_gpu, base.weight_updates, l.outputs * l.inputs);
    }
}

void sync_layer(network* nets, int n, int j)
{
    //printf("Syncing layer %d\n", j);
    int i;
    network net = nets[0];
    layer base = net.layers[j];
    cuda_set_device(net.gpu_index);
    pull_weights(base);
    for (i = 1; i < n; ++i) {
        cuda_set_device(nets[i].gpu_index);
        layer l = nets[i].layers[j];
        pull_weights(l);
        merge_weights(l, base);
    }
    scale_weights(base, 1. / n);
    for (i = 0; i < n; ++i) {
        cuda_set_device(nets[i].gpu_index);
        layer l = nets[i].layers[j];
        distribute_weights(l, base);
    }
    //printf("Done syncing layer %d\n", j);
}

typedef struct {
    network* nets;
    int n;
    int j;
} sync_args;

void* sync_layer_thread(void* ptr)
{
    sync_args args = *(sync_args*)ptr;
    sync_layer(args.nets, args.n, args.j);
    free(ptr);
    return 0;
}

float train_networks(network* nets, int n, data d, int interval)
{
    return -1;
}

float* get_network_output_layer_gpu(network net, int i)
{
    layer l = net.layers[i];
    if (l.type != REGION) cuda_pull_array(l.output_gpu, l.output, l.outputs * l.batch);
    return l.output;
}

float* get_network_output_gpu(network net)
{
    int i;
    for (i = net.n - 1; i > 0; --i) if (net.layers[i].type != COST) break;
    return get_network_output_layer_gpu(net, i);
}

float* network_predict_gpu(network net, float* input)
{
    if (net.gpu_index != cuda_get_device())
        cuda_set_device(net.gpu_index);
    int size = get_network_input_size(net) * net.batch;
    network_state state;
    state.index = 0;
    state.net = net;
    //state.input = cuda_make_array(input, size);   // memory will be allocated in the parse_network_cfg_custom()
    state.input = net.input_state_gpu;
    memcpy(net.input_pinned_cpu, input, size * sizeof(float));
    cuda_push_array(state.input, net.input_pinned_cpu, size);
    state.truth = 0;
    state.train = 0;
    state.delta = 0;
    forward_network_gpu(net, state);
    float* out = get_network_output_gpu(net);
    //cuda_free(state.input);   // will be freed in the free_network()
    return out;
}
